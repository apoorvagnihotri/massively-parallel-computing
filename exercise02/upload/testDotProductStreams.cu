// ==========================================================================
// $Id$
// ==========================================================================
// (C)opyright: 2009
//
//   Ulm University
//
// Creator: Hendrik Lensch, Holger Dammertz
// Email:   hendrik.lensch@uni-ulm.de, holger.dammertz@uni-ulm.de
// ==========================================================================
// $Log$
// ==========================================================================

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>


using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

#define MAX_BLOCKS 256
#define MAX_THREADS 128

__global__ void dotProdKernel(float* _dst, const float* _a1, const float* _a2, int _dim)
{

    // calculate how many elements each thread needs to calculate
    const unsigned int iter = _dim / (blockDim.x * gridDim.x);
    int pos = blockIdx.x * MAX_THREADS + threadIdx.x;

    // clear the output
    _dst[blockIdx.x * MAX_THREADS + threadIdx.x] = 0;

    for (int i = 0; i < iter; ++i)
    {
        _dst[blockIdx.x * MAX_THREADS + threadIdx.x] += _a1[pos] * _a2[pos];
        pos += blockDim.x * gridDim.x;
    }

    // for the last iteration, check if the elements are still available
    if (pos < _dim)
    {
        _dst[blockIdx.x * MAX_THREADS + threadIdx.x] += _a1[pos] * _a2[pos];
    }
}

/* This program sets up two large arrays of size dim and computes the
 dot product of both arrays.

 The kernel from the last exercise can be reused.
 In contrast to the last exercise the second array will be uploaded in each iteration.

 In this exercise you should implement the upload of the second array in three different ways:

 1. using simple memcopy (as in exercise 1)
 2. using memcopy from non-pageable memory (should be faster)
 3. using asynchronous memory copy with two streams

 */

int main(int argc, char* argv[])
{

    // parse command line
    int acount = 1;

    if (argc < 3)
    {
        printf("usage: testDotProductStreams <dim> <copy mode [0,1,2]>\n");
        exit(1);
    }

    // number of elements in both vectors
    int dim = atoi(argv[acount++]);

    // mode of the memory upload
    int mode = atoi(argv[acount++]);

    printf("dim: %d\n", dim);

    // Set up CPU arrays
    float* cpuOperator1[2];
    float* cpuOperator2[2];
    float* cpuResult[2];
    for (unsigned int pass = 0; pass < 2; pass++)
    {
        if (mode == 0) // simple memcpy
        {
            cpuOperator1[pass] = new float[dim];
            cpuOperator2[pass] = new float[dim];
            cpuResult[pass] = new float[MAX_THREADS * MAX_BLOCKS];
        }
        else // non-pageable memory
        {
            // !!!missing!!!
            // Allocate non-pageable memory
            hipHostMalloc((void**)&cpuOperator1[pass], dim * sizeof(float), hipHostMallocDefault); // not needed but still
            hipHostMalloc((void**)&cpuOperator2[pass], dim * sizeof(float), hipHostMallocDefault);
            hipHostMalloc((void**)&cpuResult[pass], MAX_THREADS * MAX_BLOCKS * sizeof(float), hipHostMallocDefault);
        }
    }

    // initialize the cpu arrays
    for (unsigned int pass = 0; pass < 2; pass++)
    {
        for (int i = 0; i < dim; ++i)
        {
#ifdef RTEST // With random numbers or constants...
            cpuOperator1[pass][i] = drand48();
            cpuOperator2[pass][i] = drand48();
#else
            cpuOperator1[pass][i] = 1.0f;
            cpuOperator2[pass][i] = 2.0f;
#endif
        }
    }

    // Set up the gpu arrays
    float* gpuOperator1[2];
    float* gpuOperator2[2];
    float* gpuResult[2];
    for (unsigned int pass = 0; pass < 2; pass++)
    {
        // !!! missing !!!
        hipMalloc((void**)&gpuOperator1[pass], dim * sizeof(float));
        hipMalloc((void**)&gpuOperator2[pass], dim * sizeof(float));
        hipMalloc((void**)&gpuResult[pass], MAX_THREADS* MAX_BLOCKS * sizeof(float));
    }

    // create two streams for the last mode
    hipStream_t stream[2];
    // !!! missing !!!
    // Create two streams
    hipStreamCreate(&(stream[0]));
	hipStreamCreate(&(stream[1]));

    // copy array 1 once to the device (will be static during all iterations)
    for (unsigned int pass = 0; pass < 2; pass++)
    {
        hipMemcpy(gpuOperator1[pass], cpuOperator1[pass], dim * sizeof(float), hipMemcpyHostToDevice);
    }

    // 100 Iterations for better benchmarking, we push the second array to the GPU during benchmarking
    for (int iter = 0; iter < 100; ++iter)
    {
        // Two calculations of dot products per iteration to see the advantage of streams.

        // a simplistic way of splitting the problem into threads
        dim3 blockGrid(MAX_BLOCKS);
        dim3 threadBlock(MAX_THREADS);

        switch (mode)
        {

        case 0:
            // copy a simple array
            printf("simple memcpy: \n");

            // Two passes per iteration (to be comparable to streamed version)
            for (unsigned int pass = 0; pass < 2; pass++)
            {
                hipMemcpy(gpuOperator2[pass], cpuOperator2[pass], dim * sizeof(float),
                    hipMemcpyHostToDevice);

                // call the kernel
                dotProdKernel<<<blockGrid, threadBlock>>>(gpuResult[pass], gpuOperator1[pass],
                    gpuOperator2[pass], dim);

                // download and combine the results of multiple threads
                hipMemcpy(cpuResult[pass], gpuResult[pass],
                    MAX_BLOCKS * MAX_THREADS * sizeof(float), hipMemcpyDeviceToHost);

                // Calculate the result
                float finalDotProduct = 0.0f;
                for (int i = 0; i < MAX_BLOCKS * MAX_THREADS; ++i)
                    finalDotProduct += cpuResult[pass][i];
                printf("Iteration %d, pass %d: %f\n", iter, pass, finalDotProduct);
            }

            break;

        case 1:

            // copy Array2 from pagelocked memory
            printf("pagelocked memory:\n");

            // Two passes per iteration (to be comparable to streamed version)
            for (unsigned int pass = 0; pass < 2; pass++)
            {
                // !!! missing !!!
                // Calculate the dot product with non-pageable memory.

                // we first copy the data to the GPU
                hipMemcpy(gpuOperator2[pass], cpuOperator2[pass], dim * sizeof(float),
					hipMemcpyHostToDevice);

                // we call the kernel
                dotProdKernel<<<blockGrid, threadBlock>>>(gpuResult[pass], gpuOperator1[pass],
					gpuOperator2[pass], dim);

                // we download the results
				hipMemcpy(cpuResult[pass], gpuResult[pass], MAX_BLOCKS * MAX_THREADS * sizeof(float),
                    hipMemcpyDeviceToHost);

                // Calculate the result
                float finalDotProduct = 0.0f;
                for (int i = 0; i < MAX_BLOCKS * MAX_THREADS; ++i)
                    finalDotProduct += cpuResult[pass][i];
                printf("Iteration %d, pass %d: %f\n", iter, pass, finalDotProduct);
            }

            break;

        case 2:

            // use two streams with interleaved processing
            // use asynchronous up and download
            printf("2 streams:\n");

            // !!! missing !!!
            // Calculate the dot product with streams.

            // we first copy the data to the GPU using cudaMemcpyAsync
            hipMemcpyAsync(gpuOperator2[0], cpuOperator2[0], dim * sizeof(float),
				hipMemcpyHostToDevice, stream[0]);
            hipMemcpyAsync(gpuOperator2[1], cpuOperator2[1], dim * sizeof(float),
                hipMemcpyHostToDevice, stream[1]);

            // we call the kernel using the streams
            dotProdKernel<<<blockGrid, threadBlock, 0, stream[0]>>>(gpuResult[0], gpuOperator1[0],
				gpuOperator2[0], dim);
            dotProdKernel<<<blockGrid, threadBlock, 0, stream[1]>>>(gpuResult[1], gpuOperator1[1],
                gpuOperator2[1], dim);

            // we download the results using cudaMemcpyAsync
            hipMemcpyAsync(cpuResult[0], gpuResult[0], MAX_BLOCKS * MAX_THREADS * sizeof(float),
				hipMemcpyDeviceToHost, stream[0]);
            hipMemcpyAsync(cpuResult[1], gpuResult[1], MAX_BLOCKS * MAX_THREADS * sizeof(float),
				hipMemcpyDeviceToHost, stream[1]);

            // we synchronize the streams
            hipStreamSynchronize(stream[0]);
            hipStreamSynchronize(stream[1]);

            // Calculate the result over two passes
            for (unsigned int pass = 0; pass < 2; pass++)
			{
				float finalDotProduct = 0.0f;
				for (int i = 0; i < MAX_BLOCKS * MAX_THREADS; ++i)
					finalDotProduct += cpuResult[pass][i];
				printf("Iteration %d, pass %d: %f\n", iter, pass, finalDotProduct);
			}
            break;

        } // end switch
    }

    // !!! missing !!!
    // Destroy streams
    hipStreamDestroy(stream[0]);
    hipStreamDestroy(stream[1]);

    // !!! missing !!!
    // cleanup GPU memory
    for (unsigned int pass = 0; pass < 2; pass++)
	{
		hipFree(gpuOperator1[pass]);
		hipFree(gpuOperator2[pass]);
		hipFree(gpuResult[pass]);
	}

    // cleanup host memory
    for (unsigned int pass = 0; pass < 2; pass++)
    {
        if (mode == 0) // simple memcpy
        {
            delete[] cpuOperator1[pass];
            delete[] cpuOperator2[pass];
            delete[] cpuResult[pass];
        }
        else // non-pageable memory
        {
            // !!! missing !!!
            // Free non-pageable memory
            hipHostFree(cpuOperator1[pass]);
            hipHostFree(cpuOperator2[pass]);
            hipHostFree(cpuResult[pass]);
        }
    }

    checkCUDAError("end of program");

    printf("done\n");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
