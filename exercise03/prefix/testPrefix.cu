#include "hip/hip_runtime.h"
// ==========================================================================
// $Id$
// ==========================================================================
// (C)opyright: 2009
//
//   Ulm University
//
// Creator: Hendrik Lensch, Holger Dammertz
// Email:   hendrik.lensch@uni-ulm.de, holger.dammertz@uni-ulm.de
// ==========================================================================
// $Log$
// ==========================================================================

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <vector>
#include <>

#include "PPM.hh"

using namespace std;
using namespace ppm;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

__device__ __constant__ float3 gpuClusterCol[2048];

#define THREADS 256 // this is also the size of the image (width, and h conveniently)
#define LOG_IMG_SIZE 8
#define IMG_SIZE 256
#define WINDOW 6

/* The function measures for every pixel the distance to all
 clusters, and determines the clusterID of the nearest cluster
 center. It then colors the pixel in the cluster's color.

 The cluster centers are given as an array of linear indices into
 the vector image, i.e.    _clusterInfo[0] = (x_0 + y_0 * _w).

 */
__global__ void voronoiKernel(float3 *_dst, int _w, int _h, int _nClusters, const int *_clusterInfo)
{
    // get the shared memory
    extern __shared__ int shm[];

    int nIter = _nClusters / THREADS + 1;
    // load cluster data
    for (int i = 0; i < nIter; ++i)
    {
        int pos = i * THREADS + threadIdx.x;
        if (pos < _nClusters)
        {
            shm[pos] = _clusterInfo[pos];
        }
    }

    __syncthreads();

    // compute the position within the image
    float x = blockIdx.x * blockDim.x + threadIdx.x;
    float y = blockIdx.y;

    int pos = x + y * _w;

    // determine which is the closest cluster
    float minDist = 1000000.;
    int minIdx = 0;
    for (int i = 0; i < _nClusters; ++i)
    {

        float yy = shm[i] >> LOG_IMG_SIZE;
        float xx = shm[i] % IMG_SIZE;

        float dist = (x - xx) * (x - xx) + (y - yy) * (y - yy);
        if (dist < minDist)
        {
            minDist = dist;
            minIdx = i;
        }
    }

    _dst[pos].x = gpuClusterCol[minIdx].x; // globally accessed, symbol
    _dst[pos].y = gpuClusterCol[minIdx].y;
    _dst[pos].z = gpuClusterCol[minIdx].z;

    // mark the center of each cluster
    if (minDist <= 2.)
    {
        _dst[pos].x = 255;
        _dst[pos].y = 0.;
        _dst[pos].z = 0.;
    }
}

__device__ float luminance(const float4 &_col)
{
    return 0.299 * _col.x + 0.587 * _col.y + 0.114 * _col.z;
}

/** stores a 1 in _dst if the pixel's luminance is a maximum in the
WINDOW x WINDOW neighborhood
 */
__global__ void featureKernel(int *_dst, hipTextureObject_t texImg, int _w, int _h)
{
    // compute the position within the image
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y;

    float lum = luminance(tex2D<float4>(texImg, x, y));

    bool maximum = false;

    if (lum > 20)
    {
        maximum = true;
        for (int v = y - WINDOW; v < y + WINDOW; ++v)
        {
            for (int u = x - WINDOW; u < x + WINDOW; ++u)
            {

                if (lum < luminance(tex2D<float4>(texImg, u, v)))
                {
                    maximum = false;
                }
            }
        }
    }

    if (maximum)
    {
        _dst[x + y * _w] = 1;
    }
    else
    {
        _dst[x + y * _w] = 0;
    }
}

// !!! missing !!!
// Kernels for Prefix Sum calculation (compaction, spreading, possibly shifting)
// and for generating the gpuFeatureList from the prefix sum.

__global__ void reductionKernel(int *_dst, int *_src, int _nPix) // called <<((1, 256), (256), THREADS)>>
{
    // get the thread index and the global index
    int tid = threadIdx.x;
    int index = blockIdx.y * blockDim.x + threadIdx.x;

    // store the scanline corresponding to the block in shared memory.
    __shared__ int scanline[THREADS];
    scanline[tid] = _src[index];
    __syncthreads();

    // do the reduction step
    for (int s = 1; s < blockDim.x; s *= 2)
    {
        // s is the stride, 1, 2, 4, 8, 16, ...
        if ((tid + 1) % s == 0 && tid + s < blockDim.x)
        { // the sums happen between s-1, 2s-1, 3s-1, 4s-1, ...
            scanline[tid] += scanline[tid + s];
        }
        __syncthreads();
    }

    // store the result back to global memory
    if (index < _nPix)
        _dst[index + 1] = scanline[tid];
}

__global__ void reductionKernelLast(int *_dst, int *_src, int _nPix) // called <<((1), (256), THREADS)>>
{
    // get the thread index and the global index
    int tid = threadIdx.x;
    // the last element of every row forms a scanline
    int index = threadIdx.x * blockDim.x + blockDim.x - 1;

    // store the scanline corresponding to the block in shared memory.
    __shared__ int scanline[THREADS];
    scanline[tid] = _src[index];
    __syncthreads();

    // do the reduction step
    for (int s = 1; s < blockDim.x; s *= 2)
    {
        // s is the stride, 1, 2, 4, 8, 16, ...
        if ((tid + 1) % s == 0 && tid + s < blockDim.x)
        { // the sums happen between s-1, 2s-1, 3s-1, 4s-1, ...
            scanline[tid] += scanline[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    { // since shiftedPrefixSum by definition has 0 as first element
        _dst[0] = 0;
    }

    // store the result back to global memory,
    if (index < _nPix)
        _dst[index + 1] = scanline[tid];
}

__global__ void spreadingKernel(int *_dst, int _nPix) // called <<((1, 256), (128), THREADS)>>
{
    // get the thread index and the global index
    int tid = threadIdx.x;
    int index1 = blockIdx.y * blockDim.x + threadIdx.x;
    int index2 = blockIdx.y * blockDim.x + threadIdx.x + blockDim.x;

    // store the scanline corresponding to the block in shared memory.
    __shared__ int scanline[THREADS];
    scanline[tid] = _dst[index1];
    scanline[tid + blockDim.x] = _dst[index2];
    __syncthreads();

    // do the spreading step
    // 0, 4
    // 0, 2 & 4, 6
    // 0, 1 & 2, 3 & 4, 5 & 6, 7
    // scanline is 256 everytime, that means, stride starts with 256, then 128, 64, 32, 16, 8, 4, 2, 1
    // 0, 128,
    // 0, 64 & 128, 192,
    // 0, 32 & 64, 96 & 128, 160 & 192, 224,
    for (int s = blockDim.x * 2; s > 1; s /= 2)
    {
        // s is the stride, 256, 128, 64, 32, 16, 8, 4, 2, 1
        if (tid % s == 0 && tid + s / 2 < blockDim.x * 2)
        {
            scanline[tid + s / 2] += scanline[tid];
        }
        __syncthreads();
    }

    // store the result back to global memory
    _dst[index1] = scanline[tid];
    _dst[index2] = scanline[tid + blockDim.x];
}

__global__ void spreadingKernelLast(int *_dst, int _nPix) // called <<((1), (128), THREADS)>>
{
    // get the thread index and the global index
    int tid = threadIdx.x;
    int scanLineLen = blockDim.x * 2;
    // 255 (0th row, 1st row, 2nd row, ... 127th row)
    int index1 = threadIdx.x * scanLineLen + scanLineLen - 1;
    // ... (255 + 128*256) (128th row, 129th row, 130th row, ... 255th row)
    int index2 = (threadIdx.x + blockDim.x) * scanLineLen + scanLineLen - 1;

    // store the scanline corresponding to the block in shared memory.
    __shared__ int scanline[THREADS];
    scanline[tid] = _dst[index1];
    scanline[tid + blockDim.x] = _dst[index2];
    __syncthreads();

    // do the spreading step
    for (int s = blockDim.x * 2; s > 1; s /= 2)
    {
        // s is the stride, 256, 128, 64, 32, 16, 8, 4, 2, 1
        if (tid % s == 0 && tid + s / 2 < blockDim.x * 2)
        {
            scanline[tid + s / 2] += scanline[tid];
        }
        __syncthreads();
    }

    // store the result back to global memory
    _dst[index1] = scanline[tid];
    _dst[index2] = scanline[tid + blockDim.x];
}

/* This program detects the local maxima in an image, writes their
location into a vector and then computes the Voronoi diagram of the
image given the detected local maxima as cluster centers.

A Voronoi diagram simply colors every pixel with the color of the
nearest cluster center. */

int main(int argc, char *argv[])
{

    // parse command line
    int acount = 1;
    if (argc < 4)
    {
        printf("usage: testPrefix <inImg> <outImg> <mode>\n");
        exit(1);
    }
    string inName(argv[acount++]);
    string outName(argv[acount++]);
    int mode = atoi(argv[acount++]);

    // Load the input image
    float *cpuImage;
    int w, h;
    readPPM(inName.c_str(), w, h, &cpuImage);
    int nPix = w * h;

    // Allocate GPU memory
    int *gpuFeatureImg; // Contains 1 for a feature, 0 else
    // Can be used to do the reduction step of prefix sum calculation in place
    int *gpuPrefixSumShifted; // Output buffer containing the prefix sum
    // Shifted by 1 since it contains 0 as first element by definition
    int *gpuFeatureList;   // List of pixel indices where features can be found.
    float3 *gpuVoronoiImg; // Final rgb output image
    hipMalloc((void **)&gpuFeatureImg, (nPix) * sizeof(int));

    hipMalloc((void **)&gpuPrefixSumShifted, (nPix + 1) * sizeof(int));
    hipMalloc((void **)&gpuFeatureList, 10000 * sizeof(int));

    hipMalloc((void **)&gpuVoronoiImg, nPix * 3 * sizeof(float));

    // color map for the cluster
    float clusterCol[2048 * 3];
    float *ci = clusterCol;
    for (int i = 0; i < 2048; ++i, ci += 3)
    {
        ci[0] = 32 * i % 256;
        ci[1] = (10 * i + 128) % 256;
        ci[2] = (40 * i + 255) % 256;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(gpuClusterCol), clusterCol, 2048 * 3 * sizeof(float));

    hipArray *gpuTex;
    hipChannelFormatDesc floatTex = hipCreateChannelDesc<float4>();
    hipMallocArray(&gpuTex, &floatTex, w, h);

    // pad to float4 for faster access
    float *img4 = new float[w * h * 4];

    for (int i = 0; i < w * h; ++i)
    {
        img4[4 * i] = cpuImage[3 * i];
        img4[4 * i + 1] = cpuImage[3 * i + 1];
        img4[4 * i + 2] = cpuImage[3 * i + 2];
        img4[4 * i + 3] = 0.;
    }

    // upload to array

    hipMemcpy2DToArray(gpuTex, 0, 0, img4, w * 4 * sizeof(float), w * 4 * sizeof(float), h,
                        hipMemcpyHostToDevice);

    // create texture object
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = gpuTex;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t tex = 0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr);

    cout << "setup texture" << endl;
    cout.flush();

    // calculate the block dimensions
    dim3 threadBlock(THREADS);
    dim3 blockGrid(w / THREADS, h, 1);

    printf("blockDim: %d  %d \n", threadBlock.x, threadBlock.y);
    printf("gridDim: %d  %d \n", blockGrid.x, blockGrid.y);

    featureKernel<<<blockGrid, threadBlock>>>(gpuFeatureImg, tex, w, h);

    // variable to store the number of detected features = the number of clusters
    int nFeatures;

    if (mode == 0)
    {
        ////////////////////////////////////////////////////////////
        // CPU compaction:
        ////////////////////////////////////////////////////////////

        // download result

        hipMemcpy(cpuImage, gpuFeatureImg, nPix * sizeof(float), hipMemcpyDeviceToHost);

        std::vector<int> features;

        float *ii = cpuImage;
        for (int i = 0; i < nPix; ++i, ++ii)
        {
            if (*ii > 0)
            {
                features.push_back(i);
            }
        }

        cout << "nFeatures: " << features.size() << endl;

        nFeatures = features.size();
        // upload feature vector

        hipMemcpy(gpuFeatureList, &(features[0]), nFeatures * sizeof(int), hipMemcpyHostToDevice);
    }
    else
    {
        ////////////////////////////////////////////////////////////
        // GPU compaction:
        ////////////////////////////////////////////////////////////

        // the image size is 256x256

        // !!! missing !!!
        // implement the prefixSum algorithm
        // 1. Do the reduction step for all scanlines, one scanline per block.
        dim3 gridSize(w / THREADS, h); // 1, 256
        dim3 blockSize(THREADS);       // 256
        reductionKernel<<<gridSize, blockSize, THREADS * sizeof(int)>>>(
            gpuFeatureList, gpuFeatureImg, nPix
        );

        // 2. Do the reduction step for the last elements of all scanlines, all in one block.
        gridSize = dim3(w / THREADS, 1); // 1, 1
        reductionKernelLast<<<gridSize, blockSize, THREADS * sizeof(int)>>>(
            gpuFeatureList, gpuFeatureImg, nPix
        );

        // 3. Do the spreading step for the last elements of all scanlines, all in one block.
        //    -> The last elements / elements before the scanlines have the right values now.
        blockSize = dim3(THREADS / 2); // 128
        gridSize = dim3(1, h);         // 1, 256
        spreadingKernelLast<<<gridSize, blockSize, THREADS * sizeof(int)>>>(
            gpuFeatureList, nPix
        );

        // 4. Do the spreading step for all scanlines, one scanline per block.
        gridSize = dim3(1, h / 2); // 1, 128
        spreadingKernel<<<gridSize, blockSize, THREADS * sizeof(int)>>>(
            gpuFeatureList, nPix
        );

        // !!! missing !!!
        // Make sure that gpuFeatureList is filled according to the CPU implementation
        // and that nFeatures has the correct value!
        
        // extracting the last element from the prefix sum
        hipMemcpy(&nFeatures, gpuFeatureList + nPix, sizeof(int), hipMemcpyDeviceToHost); // Replace index with the index of the value you want to extract
        cout << "nFeatures: " << nFeatures << endl;
        // upload feature vector
    }

    // now compute the Voronoi Diagram around the detected features.
    voronoiKernel<<<blockGrid, threadBlock, nFeatures * sizeof(int)>>>(gpuVoronoiImg, w, h,
                                                                       nFeatures, gpuFeatureList);

    // download final voronoi image.

    hipMemcpy(cpuImage, gpuVoronoiImg, nPix * 3 * sizeof(float), hipMemcpyDeviceToHost);
    // Write to disk
    writePPM(outName.c_str(), w, h, (float *)cpuImage);

    // Cleanup
    hipDestroyTextureObject(tex);
    hipFreeArray(gpuTex);
    hipFree(gpuFeatureList);
    hipFree(gpuFeatureImg);
    hipFree(gpuPrefixSumShifted);
    hipFree(gpuVoronoiImg);

    delete[] cpuImage;
    delete[] img4;

    checkCUDAError("end of program");

    printf("done\n");
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
