// ==========================================================================
// $Id$
// ==========================================================================
// (C)opyright: 2009-2010
//
//   Ulm University
//
// Creator: Hendrik Lensch
// Email:   {hendrik.lensch,johannes.hanika}@uni-ulm.de
// ==========================================================================
// $Log$
// ==========================================================================

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>


using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

#define MAX_BLOCKS 256
#define MAX_THREADS 128

#define RTEST // use random initialization of array

/* compute the dot product between a1 and a2. a1 and a2 are of size
 dim. The result of each thread should be stored in _dst[blockIdx.x *
 blockDim.x + threadIdx.x]. Each thread should accumulate the dot
 product of a subset of elements.
 */
__global__ void dotProdKernel(float* _dst, const float* _a1, const float* _a2, int _dim)
{

    // program your kernel here
    int x = blockIdx.x * MAX_THREADS + threadIdx.x;

    
    if (x < _dim)
    {
        float res = 0.;
        for (int j = x; j < _dim; j +=(MAX_THREADS * MAX_BLOCKS)) {
            res += _a1[j] * _a2[j];
        }
        _dst[x] = res;
    }
}

/* This program sets up two large arrays of size dim and computes the
dot product of both arrays.

The arrays are uploaded only once and the dot product is computed
multiple times. While this does not make too much sense it
demonstrated the possible speedup.  */
int main(int argc, char* argv[])
{
    // parse command line
    int acount = 1;
    
    if (argc < 3)
    {
        printf("usage: testDotProduct <dim> <GPU-flag [0,1]>\n");
        exit(1);
    }
    
    // number of elements in both vectors
    int dim = atoi(argv[acount++]);
    
    // flag indicating weather the CPU or the GPU version should be executed
    bool gpuVersion = atoi(argv[acount++]);
    //int dim = 1000000;
    //bool gpuVersion = true;
    printf("dim: %d\n", dim);

    float* cpuArray1 = new float[dim];
    float* cpuArray2 = new float[dim];

    // initialize the two arrays (either random or deterministic)
    for (int i = 0; i < dim; ++i)
    {
#ifdef RTEST
        cpuArray1[i] = rand() / (RAND_MAX + 1.0);
        cpuArray2[i] = rand() / (RAND_MAX + 1.0);
#else
        cpuArray1[i] = 2.0;
        cpuArray2[i] = i % 10;
#endif
    }

    // now the gpu stuff
    float* gpuArray1;
    float* gpuArray2;
    float* gpuResult;

    float* h;

    if (gpuVersion)
    {

        // allocate two gpuArray 1 and gpuArray 2 and gpuResult array on GPU
    	hipMalloc((void**)&gpuArray1, dim * sizeof(float));
		hipMalloc((void**)&gpuArray2, dim * sizeof(float));
        hipMalloc((void**)&gpuResult, MAX_BLOCKS * MAX_THREADS * sizeof(float));
        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!

        // copy the array once to the device
		hipMemcpy(gpuArray1, cpuArray1, dim * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(gpuArray2, cpuArray2, dim * sizeof(float), hipMemcpyHostToDevice);
        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!

        // allocate an array to download the results of all threads
        h = new float[MAX_BLOCKS * MAX_THREADS];
    }

    const int num_iters = 100;
    double finalDotProduct;

    if (!gpuVersion)
    {
        printf("cpu: ");
        for (int iter = 0; iter < num_iters; ++iter)
        {
            finalDotProduct = 0.0;
            for (int i = 0; i < dim; ++i)
            {
                finalDotProduct += cpuArray1[i] * cpuArray2[i];
            }
        }
    }
    else
    {

        // CUDA version here
        printf("gpu: ");

        // a simplistic way of splitting the problem into threads
        dim3 blockGrid(MAX_BLOCKS);
        dim3 threadBlock(MAX_THREADS);

        for (int iter = 0; iter < num_iters; ++iter)
        {
            dotProdKernel<<<blockGrid, threadBlock>>>(gpuResult, gpuArray1, gpuArray2, dim);
        }

        // download and combine the results of multiple threads on the CPU
        //cudaDeviceSynchronize();
        hipMemcpy(h, gpuResult, MAX_BLOCKS * MAX_THREADS * sizeof(float), hipMemcpyDeviceToHost);
        for (int iter = 0; iter < MAX_BLOCKS * MAX_THREADS; ++iter)
        {
            finalDotProduct = 0.0;
            for (int i = 0; i < MAX_BLOCKS * MAX_THREADS; ++i)
            {
                finalDotProduct += h[i];
            }
        }

        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!
    }

    printf("Result: %f\n", finalDotProduct);

    if (gpuVersion)
    {

        // cleanup GPU memory

        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!
        hipFree(gpuArray1);
        hipFree(gpuArray2);
        hipFree(gpuResult);

        delete[] h;
    }

    delete[] cpuArray2;
    delete[] cpuArray1;

    checkCUDAError("end of program");

    printf("done\n");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
