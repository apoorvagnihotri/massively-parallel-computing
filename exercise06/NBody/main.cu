#include "hip/hip_runtime.h"
#include "Tools.h"
#include "gltools.h"

#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <unistd.h>

#include <hip/hip_runtime.h>

using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

#define GUI
#define NUM_FRAMES 250

#define THREADS_PER_BLOCK 128
#define EPS_2 0.00001f
#define GRAVITY 0.00000001f

float randF(const float min = 0.0f, const float max = 1.0f)
{
    int randI = rand();
    float randF = (float)randI / (float)RAND_MAX;
    float result = min + randF * (max - min);

    return result;
}

inline __device__ float2 operator+(const float2 op1, const float2 op2)
{
    return make_float2(op1.x + op2.x, op1.y + op2.y);
}

inline __device__ float2 operator-(const float2 op1, const float2 op2)
{
    return make_float2(op1.x - op2.x, op1.y - op2.y);
}

inline __device__ float2 operator*(const float2 op1, const float op2)
{
    return make_float2(op1.x * op2, op1.y * op2);
}

inline __device__ float2 operator/(const float2 op1, const float op2)
{
    return make_float2(op1.x / op2, op1.y / op2);
}

inline __device__ void operator+=(float2& a, const float2 b)
{
    a.x += b.x;
    a.y += b.y;
}

int main(int argc, char** argv)
{
    if (argc != 2)
    {
        cout << "Usage: " << argv[0] << " <numBodies>" << endl;
        return 1;
    }
    unsigned int numBodies = atoi(argv[1]);
    unsigned int numBlocks = numBodies / THREADS_PER_BLOCK;
    numBodies = numBlocks * THREADS_PER_BLOCK;

    // allocate memory
    float2* hPositions = new float2[numBodies];
    float2* hVelocities = new float2[numBodies];
    float* hMasses = new float[numBodies];

    // Initialize Positions and speed
    for (unsigned int i = 0; i < numBodies; i++)
    {
        hPositions[i].x = randF(-1.0, 1.0);
        hPositions[i].y = randF(-1.0, 1.0);
        hVelocities[i].x = hPositions[i].y * 0.007f + randF(0.001f, -0.001f);
        hVelocities[i].y = -hPositions[i].x * 0.007f + randF(0.001f, -0.001f);
        hMasses[i] = randF(0.0f, 1.0f) * 10000.0f / (float)numBodies;
    }

    // TODO 1: Allocate GPU memory for
    // - Positions,
    // - Velocities,
    // - Accelerations and
    // - Masses
    // of all bodies and initialize them from the CPU arrays (where available).

    // Free host memory not needed again
    delete[] hVelocities;
    delete[] hMasses;

    // Initialize OpenGL rendering
#ifdef GUI
    initGL();
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    GLuint sp = createShaderProgram("white.vs", 0, 0, 0, "white.fs");

    GLuint vb;
    glGenBuffers(1, &vb);
    GL_CHECK_ERROR;
    glBindBuffer(GL_ARRAY_BUFFER, vb);
    GL_CHECK_ERROR;
    glBufferData(GL_ARRAY_BUFFER, sizeof(float) * 2 * numBodies, hPositions, GL_STATIC_DRAW);
    GL_CHECK_ERROR;
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    GL_CHECK_ERROR;

    GLuint va;
    glGenVertexArrays(1, &va);
    GL_CHECK_ERROR;
    glBindVertexArray(va);
    GL_CHECK_ERROR;
    glBindBuffer(GL_ARRAY_BUFFER, vb);
    GL_CHECK_ERROR;
    glEnableVertexAttribArray(glGetAttribLocation(sp, "inPosition"));
    GL_CHECK_ERROR;
    glVertexAttribPointer(glGetAttribLocation(sp, "inPosition"), 2, GL_FLOAT, GL_FALSE, 0, 0);
    GL_CHECK_ERROR;
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    GL_CHECK_ERROR;
    glBindVertexArray(0);
    GL_CHECK_ERROR;
#endif

    // Calculate
    for (unsigned int t = 0; t < NUM_FRAMES; t++)
    {
        __int64_t computeStart = continuousTimeNs();

        // TODO 3: Update accelerations of all bodies here.

        // TODO 4: Update velocities and positions of all bodies here.

        hipDeviceSynchronize();
        cout << "Frame compute time: " << (continuousTimeNs() - computeStart) << "ns" << endl;

        // TODO 5: Download the updated positions into the hPositions array for rendering.

#ifdef GUI
        // Upload positions to OpenGL
        glBindBuffer(GL_ARRAY_BUFFER, vb);
        GL_CHECK_ERROR;
        glBufferData(GL_ARRAY_BUFFER, sizeof(float) * 2 * numBodies, hPositions, GL_STATIC_DRAW);
        GL_CHECK_ERROR;
        glBindBuffer(GL_ARRAY_BUFFER, 0);
        GL_CHECK_ERROR;

        // Draw
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        GL_CHECK_ERROR;
        glUseProgram(sp);
        GL_CHECK_ERROR;
        glBindVertexArray(va);
        GL_CHECK_ERROR;
        glDrawArrays(GL_POINTS, 0, numBodies);
        GL_CHECK_ERROR;
        glBindVertexArray(0);
        GL_CHECK_ERROR;
        glUseProgram(0);
        GL_CHECK_ERROR;
        swapBuffers();
#endif
    }

#ifdef GUI
    cout << "Done." << endl;
    sleep(2);
#endif

    // Clean up
#ifdef GUI
    glDeleteProgram(sp);
    GL_CHECK_ERROR;
    glDeleteVertexArrays(1, &va);
    GL_CHECK_ERROR;
    glDeleteBuffers(1, &vb);
    GL_CHECK_ERROR;

    glDeleteProgram(sp);
    exitGL();
#endif

    // TODO 2: Clean up your allocated memory

    delete[] hPositions;

    checkCUDAError("end of program");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
